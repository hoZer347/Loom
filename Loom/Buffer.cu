#include "hip/hip_runtime.h"
#include "Buffer.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>


namespace Loom
{
	__device__ unsigned char* buffer;

	__global__ static void __DrawRect(
		void* output,
		const unsigned int* position,
		const unsigned int* buffer_dims)
	{
		((unsigned char*)output)[threadIdx.x + 0] = 255;
		((unsigned char*)output)[threadIdx.x + 1] = 0;
		((unsigned char*)output)[threadIdx.x + 2] = 0;
		((unsigned char*)output)[threadIdx.x + 3] = 255;
	};

	void DrawRect(
		void* output,
		const unsigned int* position,
		const unsigned int* dimensions,
		const unsigned int* buffer_dims)
	{
		__DrawRect<<<{ dimensions[0], dimensions[1] }, 1>>>(
			output,
			position,
			buffer_dims);
	};

	void BufferSet(unsigned char* buffer_ptr)
	{
		buffer = buffer_ptr;
	};

	void BufferMalloc(void* ptr, size_t size) noexcept
	{
		if (hipDeviceSynchronize())
			std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

		if (hipMalloc(&ptr, size))
			std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	};

	void BufferSend(void* dst, const void* src, size_t size) noexcept
	{
		if (hipDeviceSynchronize())
			std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

		if (hipMemcpy(dst, src, size, hipMemcpyHostToDevice))
			std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	};

	void BufferCopy(void* dst, const void* src, size_t size) noexcept
	{
		if (hipDeviceSynchronize())
			std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

		if (hipMemcpy(dst, src, size, hipMemcpyDeviceToHost))
			std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	};

	void BufferFree(void* ptr) noexcept
	{
		if (hipDeviceSynchronize())
			std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

		if (hipFree(ptr))
			std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	};

	void BufferSync() noexcept
	{
		if (hipDeviceSynchronize())
			std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	};
};
