#include "hip/hip_runtime.h"
#include "Geometry.cuh"

#include <hip/hip_runtime.h>
#include <>

#include <iostream>


namespace Loom
{
	__global__ void __DrawRect(
		unsigned char* buffer_data,
		const unsigned int buffer_w,
		const unsigned int buffer_h,
		const unsigned int rect_x,
		const unsigned int rect_y,
		const unsigned int rect_w,
		const unsigned int rect_h)
	{
		// Calculate the global thread index
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

		// Ensure the thread is within the buffer bounds
		if (x >= buffer_w || y >= buffer_h) return;

		// Check if the pixel is inside the rectangle
		if (x >= rect_x &&
			x < rect_x + rect_w &&
			y >= rect_y &&
			y < rect_y + rect_h)
		{
			unsigned int index = (y * buffer_w + x) * 4; // Assuming 4 bytes per pixel (RGBA)

			// Set the pixel color
			buffer_data[index]		= 128;
			buffer_data[index + 1]	= 0;
			buffer_data[index + 2]	= 0;
			buffer_data[index + 3]	= 255;
		};
	};

	void DrawRectToBuffer(
		void* buffer_data,
		const unsigned int& buffer_w,
		const unsigned int& buffer_h,
		const unsigned int& rect_x,
		const unsigned int& rect_y,
		const unsigned int& rect_w,
		const unsigned int& rect_h)
	{
		dim3 blockSize(16, 16); // 16x16 threads per block
		dim3 gridSize(	(buffer_w + blockSize.x - 1) / blockSize.x,
						(buffer_h + blockSize.y - 1) / blockSize.y);

		__DrawRect<<<gridSize, blockSize>>>(
			(unsigned char*)buffer_data,
			buffer_w,
			buffer_h,
			rect_x,
			rect_y,
			rect_w,
			rect_h);
	};
};
